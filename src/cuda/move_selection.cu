#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include "cuda/move_selection.cuh"

namespace checkers::gpu::move_selection
{

__device__ __forceinline__ void RandomSelection(
    const u64 board_idx,
    // Moves
    const move_t* d_moves, const u8* d_move_counts,
    // Seeds
    const u8* seeds,
    // Output
    move_t* d_best_moves
)
{
    // We'll store invalid if no moves are found.
    move_t chosen_move = MoveConstants::kInvalidMove;

    const u64 board_fields_begin = board_idx * BoardConstants::kBoardSize;

    // Start offset in the piece array is chosen by the seed.
    const board_index_t initial_figure_idx = seeds[board_idx] % BoardConstants::kBoardSize;

    // First, find any piece with at least 1 move, searching in a wrap-around manner.
    board_index_t chosen_figure_idx = BoardConstants::kBoardSize;  // sentinel (means not found)
    for (board_index_t i = 0; i < BoardConstants::kBoardSize; i++) {
        board_index_t candidate = (initial_figure_idx + i) % BoardConstants::kBoardSize;
        u8 count_for_candidate  = d_move_counts[board_fields_begin + candidate];
        if (count_for_candidate > 0) {
            chosen_figure_idx = candidate;
            break;
        }
    }

    // If a piece with moves was found:
    if (chosen_figure_idx < BoardConstants::kBoardSize) {
        // Number of valid moves for that piece
        u8 num_piece_moves = d_move_counts[board_fields_begin + chosen_figure_idx];

        // Pick a random sub-move from 0..(num_piece_moves-1)
        u8 random_sub_move_idx = seeds[board_idx] % num_piece_moves;

        // Now compute the flattened index into d_moves:
        const u64 moves_base     = board_idx * (BoardConstants::kBoardSize * move_gen::kNumMaxMovesPerPiece);
        const u64 piece_base     = chosen_figure_idx * move_gen::kNumMaxMovesPerPiece;
        const u64 final_move_idx = moves_base + piece_base + random_sub_move_idx;

        chosen_move = d_moves[final_move_idx];
    }

    d_best_moves[board_idx] = chosen_move;
}
__device__ __forceinline__ void SelectBestMovesForBoardIdx(
    const board_index_t board_idx, const u32* d_whites, const u32* d_blacks, const u32* d_kings, const move_t* d_moves,
    const u8* d_move_counts, const move_flags_t* d_move_capture_mask, const move_flags_t* d_per_board_flags,
    const u64 n_boards, const u8* seeds, move_t* d_best_moves
)
{
    RandomSelection(board_idx, d_moves, d_move_counts, seeds, d_best_moves);
}

__global__ void SelectBestMoves(
    // Board states (unused in random selection, but placeholders for expansions)
    const u32* d_whites, const u32* d_blacks, const u32* d_kings,
    // Moves
    const move_t* d_moves, const u8* d_move_counts, const move_flags_t* d_move_capture_mask,
    const move_flags_t* d_per_board_flags,
    // Number of boards
    const u64 n_boards,
    // Seeds
    const u8* seeds,
    // Output
    move_t* d_best_moves
)
{
    for (u64 board_idx = blockIdx.x * blockDim.x + threadIdx.x; board_idx < n_boards;
         board_idx += gridDim.x * blockDim.x) {
        SelectBestMovesForBoardIdx(
            board_idx, d_whites, d_blacks, d_kings, d_moves, d_move_counts, d_move_capture_mask, d_per_board_flags,
            n_boards, seeds, d_best_moves
        );
    }
}

}  // namespace checkers::gpu::move_selection
