#include "hip/hip_runtime.h"
#include "cuda/apply_move.cuh"
#include "cuda/board_helpers.cuh"
#include "cuda/capture_lookup_table.cuh"
#include "iostream"

namespace checkers::gpu::apply_move
{
__device__ void ApplyMoveOnBoardIdx(
    const board_index_t board_idx,
    // Board States
    board_t *d_whites, board_t *d_blacks, board_t *d_kings,
    // Moves
    const move_t *d_moves,
    // Number of boards to process
    const u64 n_boards
)
{
    board_index_t from = move_gen::DecodeMove<move_gen::MovePart::From>(d_moves[board_idx]);
    board_index_t to   = move_gen::DecodeMove<move_gen::MovePart::To>(d_moves[board_idx]);

    // board[from] = board[to]
    d_whites[board_idx] |= ((d_whites[board_idx] >> from) & 1) << to;
    d_blacks[board_idx] |= ((d_blacks[board_idx] >> from) & 1) << to;
    d_kings[board_idx] |= ((d_kings[board_idx] >> from) & 1) << to;

    // board[from] = 0
    d_whites[board_idx] &= ~(1 << from);
    d_blacks[board_idx] &= ~(1 << from);
    d_kings[board_idx] &= ~(1 << from);

    // 0 out everything in between the move
    board_t kCaptureMask = d_kCaptureLookUpTable[from * BoardConstants::kBoardSize + to];
    d_blacks[board_idx] &= kCaptureMask;
    d_whites[board_idx] &= kCaptureMask;
    d_kings[board_idx] &= kCaptureMask;
}

__global__ void ApplyMove(
    // Board States
    board_t *d_whites, board_t *d_blacks, board_t *d_kings,
    // Moves
    const move_t *d_moves,
    // Number of boards to process
    const u64 n_boards
)
{
    u64 board_idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (; board_idx < n_boards; board_idx += gridDim.x * blockDim.x) {
        ApplyMoveOnBoardIdx(board_idx, d_whites, d_blacks, d_kings, d_moves, n_boards);
    }
}
}  // namespace checkers::gpu::apply_move
