#include "cpu/capture_lookup_table.hpp"
#include "cuda/apply_move.cuh"
#include "cuda/board_helpers.cuh"
#include "cuda/capture_lookup_table.cuh"
#include "cuda/cuda_utils.cuh"

namespace checkers::gpu::apply_move
{
__constant__ board_t d_kCaptureLookUpTable[BoardConstants::kBoardSize * BoardConstants::kBoardSize];

void InitializeCaptureLookupTable()
{
    // Flatten the 2D host array to a 1D array
    std::array<board_t, BoardConstants::kBoardSize * BoardConstants::kBoardSize> flatTable{};
    for (size_t i = 0; i < BoardConstants::kBoardSize; ++i) {
        for (size_t j = 0; j < BoardConstants::kBoardSize; ++j) {
            flatTable[i * BoardConstants::kBoardSize + j] = checkers::cpu::apply_move::h_kCaptureLookUpTable[i][j];
        }
    }

    // Copy the flattened data to constant memory on the device
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(
        d_kCaptureLookUpTable), flatTable.data(),
        sizeof(board_t) * BoardConstants::kBoardSize * BoardConstants::kBoardSize
    ));
}
}  // namespace checkers::gpu::apply_move
